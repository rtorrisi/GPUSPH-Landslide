#include "hip/hip_runtime.h"
#include <string>

#include "ChuteLandslide.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

ChuteLandslide::ChuteLandslide(GlobalData *_gdata) : XProblem(_gdata)
{
	m_name = "ChuteLandslide";

	setFramework();

	const int mlsIters = get_option("mls", 0); // --mls N to enable MLS filter every N iterations
	const int ppH = get_option("ppH", 16); // --ppH N to change deltap to H/N

	if (mlsIters > 0) addFilter(MLS_FILTER, mlsIters);

	set_deltap(m_cupInitialHeight/ppH);

	m_chuteThickness = 3*m_deltap;
	m_chuteObliqueDelta = sqrt(2) * m_chuteThickness;
	m_chuteObliqueLength += m_chuteObliqueDelta;

	// SPH parameters
	setSPHParameters();

	// Physical parameters
	setPhysicalParameters();

	// Drawing and saving times
	add_writer(VTKWRITER, 0.01);

	// Building the geometry
	buildGeometry();
}

void ChuteLandslide::setFramework()
{
	// density diffusion terms, see DensityDiffusionType
	const int	rhodiff = get_option("density-diffusion", 1);
	const bool	newtonian = get_option("newtonian", false);

	SETUP_FRAMEWORK(
		rheology<PAPANASTASIOU>,
		turbulence_model<LAMINAR_FLOW>,
		visc_model<MORRIS>,
		visc_average<HARMONIC>,
		computational_visc<DYNAMIC>,
		boundary<DYN_BOUNDARY>
	).select_options(
		newtonian, rheology<NEWTONIAN>()
	);
}

void ChuteLandslide::setSPHParameters()
{
	simparams()->buildneibsfreq = 10;
	simparams()->tend = 5.0f;
}

void ChuteLandslide::setPhysicalParameters()
{
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	const float g = length(physparams()->gravity);
	const float maxvel = sqrt(2*g*1);
	// purely for cosmetic reason, let's round the soundspeed to the next integer
	const float c0 = ceil(10*maxvel);
	add_fluid(1000.0);
	set_equation_of_state(0, 7.0f, c0);
	physparams()->dcoeff = 5.0f*g*1;
	physparams()->r0 = m_deltap;
	//physparams()->visccoeff = 0.05f;
	set_kinematic_visc(0, 3.0e-2f);
	if (YIELDING_RHEOLOGY(simparams()->rheologytype))
		set_yield_strength(0, 1.0f);
}

void ChuteLandslide::buildGeometry()
{
	setPositioning(PP_CENTER);

	double3 maxChutePoint = make_double3( 0, //x
		- m_chuteObliqueLength*cos(m_chuteInclinationAngle), //y
		m_chuteObliqueLength*sin(m_chuteInclinationAngle) //z
	);

	double3 cupPositionDelta = make_double3( 0, //x
		(m_sphereRadius*cos(m_cupLatitudeCutAngle))*cos(m_chuteInclinationAngle), //y
		- (m_sphereRadius*cos(m_cupLatitudeCutAngle))*sin(m_chuteInclinationAngle) //z
	);

	double3 cupLatitudeCutDelta = make_double3( 0, //x
		- (m_sphereRadius-m_cupInitialHeight)*cos(m_chuteInclinationAngle), //y
		- (m_sphereRadius-m_cupInitialHeight)*sin(m_chuteInclinationAngle) //z
	);

	Point cupOriginPoint = Point(
		m_chuteWidth / 2.f,
		maxChutePoint.y + cupPositionDelta.y + cupLatitudeCutDelta.y,
		maxChutePoint.z + cupPositionDelta.z + cupLatitudeCutDelta.z
	);

	GeometryID fluid = addSphere(GT_FLUID, FT_SOLID, cupOriginPoint, m_sphereRadius);
	rotate(fluid, m_chuteInclinationAngle, 0.f, 0.f);

	GeometryID obliquePlane = addPlane(0.0f, 1.0f, 1.0f, 0.0f, FT_UNFILL);
	setIntersectionType(obliquePlane, IT_INTERSECT);

	setPositioning(PP_CORNER);

	GeometryID horizontalBox = addBox(GT_FIXED_BOUNDARY, FT_SOLID,
		Point(0, m_chuteHorizontalOffset, 0), m_chuteWidth, m_chuteHorizontalLength, m_chuteThickness);

	GeometryID obliqueBox = addBox(GT_FIXED_BOUNDARY, FT_SOLID,
		Point(0,0,0), m_chuteWidth, m_chuteObliqueLength, m_chuteThickness);
	rotate(obliqueBox, M_PI+m_chuteInclinationAngle, 0.f, 0.f);

	GeometryID horizontalPlane = addPlane(0.f, 0.f, 1.f, 0.01f, FT_UNFILL);
	setIntersectionType(horizontalPlane, IT_INTERSECT);

	// boolean value to add a "Tree" (cylinder) in the middle of the chute
	const bool addTree = get_option("tree", false);
	
	if(addTree)
	{
		GeometryID tree = addCylinder(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_chuteWidth/2.f - 0.1f, -0.4f, 0), 0.1f, 1.f);
	}
}
