#include "hip/hip_runtime.h"
#include "Landslide.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

Landslide::Landslide(GlobalData *_gdata) : XProblem(_gdata)
{
	m_name = "Landslide";

	setFramework();

	const int mlsIters = get_option("mls", 0); // --mls N to enable MLS filter every N iterations
	const int ppH = get_option("ppH", 18); // --ppH N to change deltap to H/N

	if (mlsIters > 0) addFilter(MLS_FILTER, mlsIters);
	set_deltap(1.f/ppH);

	// SPH parameters
	setSPHParameters();

	// Physical parameters
	setPhysicalParameters();

	// Drawing and saving times
	add_writer(VTKWRITER, 1.0);

	// Building the geometry
	buildGeometry();
}

void Landslide::setFramework()
{
	// density diffusion terms, see DensityDiffusionType
	const int	rhodiff = get_option("density-diffusion", 1);

	SETUP_FRAMEWORK(
		viscosity<DYNAMICVISC>,
		boundary<DYN_BOUNDARY>,
		add_flags<ENABLE_PLANES>
	);
}

void Landslide::setSPHParameters()
{
	simparams()->buildneibsfreq = 10;
}

void Landslide::setPhysicalParameters()
{
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	const float g = length(physparams()->gravity);
	const float maxvel = sqrt(2*g*1);
	// purely for cosmetic reason, let's round the soundspeed to the next integer
	const float c0 = ceil(10*maxvel);
	add_fluid(1000.0);
	set_equation_of_state(0, 7.0f, c0);
	physparams()->dcoeff = 5.0f*g*1;
	physparams()->r0 = m_deltap;
	//physparams()->visccoeff = 0.05f;
	set_kinematic_visc(0, 3.0e-2f);
	//set_kinematic_visc(0, 1.0e-6f);
	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;
	physparams()->epsxsph = 0.5f;
}

void Landslide::buildGeometry()
{
	setPositioning(PP_CENTER);

	GeometryID fluid = addSphere(GT_FLUID, FT_SOLID,
		Point(m_chuteWidth / 2, -1.105, 1.105), m_fluidRadius);
	rotate(fluid, m_chuteInclinationAngle, 0.f, 0.f);

	setPositioning(PP_CORNER);

	GeometryID horizontalBox = addBox(GT_FIXED_BOUNDARY, FT_SOLID,
		Point(0, m_offset, 0), m_chuteWidth, m_chuteHorizontalLength, m_chuteThickness);

	GeometryID obliquePlane = addPlane(0.0f, 1.0f, 1.0f, 0.0f);
	setIntersectionType(obliquePlane, IT_INTERSECT);
	deleteGeometry(obliquePlane);
	
	GeometryID obliqueBox = addBox(GT_FIXED_BOUNDARY, FT_SOLID,
		Point(0,0,0), m_chuteWidth, m_chuteObliqueLength + m_deltaChuteObliqueLength, m_chuteThickness);
	rotate(obliqueBox, m_chuteInclinationAngle, 0.f, 0.f);

	GeometryID horizontalPlane = addPlane(0.0f, 0.0f, 1.0f, 0.1f);
	setIntersectionType(horizontalPlane, IT_INTERSECT);
	deleteGeometry(horizontalPlane);
}