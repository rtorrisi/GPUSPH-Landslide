#include "hip/hip_runtime.h"
#include "Landslide.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

Landslide::Landslide(GlobalData *_gdata) : XProblem(_gdata)
{
	m_name = "Landslide";

	setFramework();

	const int mlsIters = get_option("mls", 0); // --mls N to enable MLS filter every N iterations
	const int ppH = get_option("ppH", 16); // --ppH N to change deltap to H/N

	if (mlsIters > 0) addFilter(MLS_FILTER, mlsIters);

	set_deltap(m_cupInitialHeight/ppH);
	
	m_chuteThickness = 3*m_deltap;
	m_chuteObliqueDelta = sqrt(2) * m_chuteThickness;
	m_chuteObliqueLength += m_chuteObliqueDelta;

	// SPH parameters
	setSPHParameters();

	// Physical parameters
	setPhysicalParameters();

	// Drawing and saving times
	add_writer(VTKWRITER, 0.01);

	// Building the geometry
	buildGeometry();
}

void Landslide::setFramework()
{
	// density diffusion terms, see DensityDiffusionType
	const int	rhodiff = get_option("density-diffusion", 1);

	SETUP_FRAMEWORK(
		viscosity<DYNAMICVISC>,
		boundary<DYN_BOUNDARY>,
		add_flags<ENABLE_PLANES>
	);
}

void Landslide::setSPHParameters()
{
	simparams()->buildneibsfreq = 10;
}

void Landslide::setPhysicalParameters()
{
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	const float g = length(physparams()->gravity);
	const float maxvel = sqrt(2*g*1);
	// purely for cosmetic reason, let's round the soundspeed to the next integer
	const float c0 = ceil(10*maxvel);
	add_fluid(1000.0);
	set_equation_of_state(0, 7.0f, c0);
	physparams()->dcoeff = 5.0f*g*1;
	physparams()->r0 = m_deltap;
	//physparams()->visccoeff = 0.05f;
	set_kinematic_visc(0, 3.0e-2f);
	//set_kinematic_visc(0, 1.0e-6f);
	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;
	physparams()->epsxsph = 0.5f;
}

void Landslide::buildGeometry()
{
	setPositioning(PP_CENTER);

	double3 maxChutePoint = make_double3( 0, //x
		- m_chuteObliqueLength*cos(m_chuteInclinationAngle), //y
		m_chuteObliqueLength*sin(m_chuteInclinationAngle) //z
	);

	double3 cupPositionDelta = make_double3( 0, //x
		(m_sphereRadius*cos(m_cupLatitudeCutAngle))*cos(m_chuteInclinationAngle), //y
		- (m_sphereRadius*cos(m_cupLatitudeCutAngle))*sin(m_chuteInclinationAngle) //z
	);

	double3 cupLatitudeCutDelta = make_double3( 0, //x
		- (m_sphereRadius-m_cupInitialHeight)*cos(m_chuteInclinationAngle), //y
		- (m_sphereRadius-m_cupInitialHeight)*sin(m_chuteInclinationAngle) //z
	);

	Point cupOriginPoint = Point(
		m_chuteWidth / 2.f,
		maxChutePoint.y + cupPositionDelta.y + cupLatitudeCutDelta.y,
		maxChutePoint.z + cupPositionDelta.z + cupLatitudeCutDelta.z
	);

	GeometryID fluid = addSphere(GT_FLUID, FT_SOLID, cupOriginPoint, m_sphereRadius);
	rotate(fluid, m_chuteInclinationAngle, 0.f, 0.f);

	GeometryID obliquePlane = addPlane(0.0f, 1.0f, 1.0f, 0.0f);
	setIntersectionType(obliquePlane, IT_INTERSECT);
	//deleteGeometry(obliquePlane);

	setPositioning(PP_CORNER);

	GeometryID horizontalBox = addBox(GT_FIXED_BOUNDARY, FT_SOLID,
		Point(0, m_chuteHorizontalOffset, 0), m_chuteWidth, m_chuteHorizontalLength, m_chuteThickness);

	GeometryID obliqueBox = addBox(GT_FIXED_BOUNDARY, FT_SOLID,
		Point(0,0,0), m_chuteWidth, m_chuteObliqueLength, m_chuteThickness);
	rotate(obliqueBox, M_PI+m_chuteInclinationAngle, 0.f, 0.f);

	GeometryID horizontalPlane = addPlane(0.f, 0.f, 1.f, 0.01f);
	setIntersectionType(horizontalPlane, IT_INTERSECT);
	//deleteGeometry(horizontalPlane);
}