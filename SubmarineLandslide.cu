#include "hip/hip_runtime.h"
#include <string>

#include "SubmarineLandslide.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

SubmarineLandslide::SubmarineLandslide(GlobalData *_gdata) : XProblem(_gdata)
{
	m_name = "SubmarineLandslide";

	setFramework();

	const int mlsIters = get_option("mls", 0); // --mls N to enable MLS filter every N iterations
	const int ppH = get_option("ppH", 32); // --ppH N to change deltap to H/N

	if (mlsIters > 0) addFilter(MLS_FILTER, mlsIters);

	set_deltap(m_bulkHeight/ppH);

	// SPH parameters
	setSPHParameters();

	// Physical parameters
	setPhysicalParameters();

	// Drawing and saving times
	add_writer(VTKWRITER, 0.01);

	// Building the geometry
	buildGeometry();
}

void SubmarineLandslide::setFramework()
{
	// density diffusion terms, see DensityDiffusionType
	const int	rhodiff = get_option("density-diffusion", 1);
	const bool	newtonian = get_option("newtonian", false);

	SETUP_FRAMEWORK(
		rheology<PAPANASTASIOU>,
		turbulence_model<LAMINAR_FLOW>,
		visc_model<MORRIS>,
		visc_average<HARMONIC>,
		computational_visc<DYNAMIC>,
		boundary<DYN_BOUNDARY>,
		add_flags<ENABLE_PLANES>
	).select_options(
		newtonian, rheology<NEWTONIAN>()
	);
}

void SubmarineLandslide::setSPHParameters()
{
	simparams()->buildneibsfreq = 10;
	simparams()->tend = 5.0f;
}

void SubmarineLandslide::setPhysicalParameters()
{
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	const float g = length(physparams()->gravity);
	const float maxvel = sqrt(2*g*1);
	// purely for cosmetic reason, let's round the soundspeed to the next integer
	const float c0 = ceil(10*maxvel);
	add_fluid(1000.0);
	set_equation_of_state(0, 7.0f, c0);
	physparams()->dcoeff = 5.0f*g*1;
	physparams()->r0 = m_deltap;
	//physparams()->visccoeff = 0.05f;
	set_kinematic_visc(0, 3.0e-2f);
	if (YIELDING_RHEOLOGY(simparams()->rheologytype))
		set_yield_strength(0, 1.0f);
}

void SubmarineLandslide::buildGeometry()
{
	setPositioning(PP_CORNER);

	GeometryID obliqueChute = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(0.f, 0.f, 0.f),
		m_chuteWidth, m_depth, m_chuteHeight);

	// This plane is used to erase unnecessary fixed boundary particles
	GeometryID optimizationPlane = addPlane(1.f, 0.f, 1.f, -m_chuteWidth + m_chuteThickness, FT_UNFILL);
		setIntersectionType(optimizationPlane, IT_INTERSECT);

	GeometryID horizontalChute = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(0.f, 0.f, 0.f),
		m_chuteUpperWidth - m_gapFixOffset2, m_depth, m_chuteHeight);

	// This box is used to erase unnecessary fixed boundary particles
	GeometryID optimizationBox = addBox(GT_FIXED_BOUNDARY, FT_NOFILL, Point(0.f, 0.f, 0.f),
		m_chuteUpperWidth - m_gapFixOffset2, m_depth, m_chuteHeight - m_chuteThickness);

	GeometryID obliquePlane = addPlane(1.f, 0.f, 1.f, -m_chuteWidth - m_gapFixOffset, FT_UNFILL);
	setIntersectionType(obliquePlane, IT_SUBTRACT);
	
	GeometryID waterBox = addBox(GT_FLUID, FT_SOLID, Point(0.f, 0.f, 0.f),
		m_waterBoxWidth, m_depth, m_waterBoxHeight);
	
	Point bulkPoint = Point(m_chuteUpperWidth, 0.f, m_chuteHeight - m_bulkHeight);
	GeometryID bulk = addBox(GT_FLUID, FT_SOLID, bulkPoint,
		m_bulkWidth, m_depth, m_bulkHeight);
	
	GeometryID obliquePlane2 = addPlane(1.f, 0.f, 1.f, -m_chuteWidth + m_gapFixOffset, FT_UNFILL);
	setEraseOperation(obliquePlane2, ET_ERASE_FLUID);
	setIntersectionType(obliquePlane2, IT_INTERSECT);

	GeometryID fixWaterBox = addBox(GT_FLUID, FT_SOLID, Point(0.f, 0.f, m_chuteHeight + 0.025),
		m_waterBoxWidth, m_depth, m_waterBoxHeight - m_chuteHeight - m_gapFixOffset);
}
