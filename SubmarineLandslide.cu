#include "hip/hip_runtime.h"
#include <string>

#include "SubmarineLandslide.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

SubmarineLandslide::SubmarineLandslide(GlobalData *_gdata) : XProblem(_gdata)
{
	m_name = "SubmarineLandslide";

	setFramework();

	const int mlsIters = get_option("mls", 0); // --mls N to enable MLS filter every N iterations
	const int ppH = get_option("ppH", 32); // --ppH N to change deltap to H/N

	if (mlsIters > 0) addFilter(MLS_FILTER, mlsIters);

	set_deltap(m_bulkHeight/ppH);

	// SPH parameters
	setSPHParameters();

	// Physical parameters
	setPhysicalParameters();

	// Drawing and saving times
	add_writer(VTKWRITER, 0.01);

	// Building the geometry
	buildGeometry();
}

void SubmarineLandslide::setFramework()
{
	// density diffusion terms, see DensityDiffusionType
	const int	rhodiff = get_option("density-diffusion", 1);
	const bool	newtonian = get_option("newtonian", false);

	SETUP_FRAMEWORK(
		rheology<PAPANASTASIOU>,
		turbulence_model<LAMINAR_FLOW>,
		visc_model<MORRIS>,
		visc_average<HARMONIC>,
		computational_visc<DYNAMIC>,
		boundary<DYN_BOUNDARY>,
		periodicity<PERIODIC_Y>
	).select_options(
		newtonian, rheology<NEWTONIAN>()
	);
}

void SubmarineLandslide::setSPHParameters()
{
	simparams()->buildneibsfreq = 10;
	simparams()->tend = 5.0f;
}

void SubmarineLandslide::setPhysicalParameters()
{
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	const float g = length(physparams()->gravity);
	const float maxvel = sqrt(2*g*1);
	// purely for cosmetic reason, let's round the soundspeed to the next integer
	const float c0 = ceil(10*maxvel);
	add_fluid(1000.0);
	set_equation_of_state(0, 7.0f, c0);
	physparams()->dcoeff = 5.0f*g*1;
	physparams()->r0 = m_deltap;
	//physparams()->visccoeff = 0.05f;
	set_kinematic_visc(0, 3.0e-2f);
	if (YIELDING_RHEOLOGY(simparams()->rheologytype))
		set_yield_strength(0, 1.0f);
}

void SubmarineLandslide::buildGeometry()
{
	setPositioning(PP_CORNER);

	GeometryID obliqueChute = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(0.f, 0.f, 0.f),
		m_chuteWidth, m_depth, m_chuteHeight);

	// This plane is used to erase unnecessary fixed boundary particles
	GeometryID optimizationPlane = addPlane(1.f, 0.f, 1.f, -m_chuteWidth + m_chuteThickness, FT_UNFILL);
		setIntersectionType(optimizationPlane, IT_INTERSECT);

	GeometryID horizontalChute = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(0.f, 0.f, 0.f),
		m_chuteUpperWidth - m_gapFixOffset2, m_depth, m_chuteHeight);

	// This box is used to erase unnecessary fixed boundary particles
	GeometryID optimizationBox = addBox(GT_FIXED_BOUNDARY, FT_NOFILL, Point(0.f, 0.f, 0.f),
		m_chuteUpperWidth - m_gapFixOffset2, m_depth, m_chuteHeight - m_chuteThickness + 0.025f);

	GeometryID obliquePlane = addPlane(1.f, 0.f, 1.f, -m_chuteWidth - m_gapFixOffset, FT_UNFILL);
	setIntersectionType(obliquePlane, IT_SUBTRACT);

	GeometryID baseBoundary = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_chuteWidth - 0.1f, 0.f, -0.125f),
		m_waterBoxWidth - m_chuteWidth + 0.2f, m_depth, 0.1f);

	GeometryID leftBoundary = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(-0.125f, 0.f, m_chuteHeight - 0.1f),
		0.1f, m_depth, 0.2f);
	
	GeometryID rightBoundary = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_waterBoxWidth + 0.025f, 0.f, 0.f),
		0.1f, m_depth, m_waterBoxHeight);

	// boolean value to fill the chute with with water
	const int fillWater = get_option("fillWater", true);

	if(fillWater)
	{
		GeometryID waterBox = addBox(GT_FLUID, FT_SOLID, Point(0.f, 0.f, 0.f),
			m_waterBoxWidth, m_depth, m_waterBoxHeight);
	}
	
	Point bulkPoint = Point(m_chuteUpperWidth, 0.f, m_chuteHeight - m_bulkHeight);
	GeometryID bulk = addBox(GT_FLUID, FT_SOLID, bulkPoint,
		m_bulkWidth, m_depth, m_bulkHeight);
	
	GeometryID obliquePlane2 = addPlane(1.f, 0.f, 1.f, -m_chuteWidth + m_gapFixOffset, FT_UNFILL);
	setEraseOperation(obliquePlane2, ET_ERASE_FLUID);
	setIntersectionType(obliquePlane2, IT_INTERSECT);

	if(fillWater)
	{
		GeometryID fixWaterBox = addBox(GT_FLUID, FT_SOLID, Point(0.f, 0.f, m_chuteHeight + 0.025),
			m_waterBoxWidth, m_depth, m_waterBoxHeight - m_chuteHeight - m_gapFixOffset);
	}

	setPositioning(PP_CENTER);

	// boolean value to a "Tree" (cylinder) in the middle of the chute
	const int addTree = get_option("tree", false);
	
	if(addTree){
		GeometryID tree = addCylinder(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_chuteWidth, m_depth/2.f, m_waterBoxHeight/2.f), 0.1f, m_waterBoxHeight);
	}

	// boolean value to add three "Pillars" (cylinder) at the end of the chute
	const int addPillars = get_option("pillars", false);

	if(addPillars)
	{
		GeometryID leftPillar = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_chuteWidth + 1.f, m_depth/3.f, m_waterBoxHeight/2.f), 0.1f, 0.1f, m_waterBoxHeight);
		GeometryID rightPillar = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_chuteWidth + 1.f, 2*m_depth/3.f, m_waterBoxHeight/2.f), 0.1f, 0.1f, m_waterBoxHeight);
	}

}
