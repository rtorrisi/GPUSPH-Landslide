#include "hip/hip_runtime.h"
#include <string>

#include "SubmarineLandslide.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

SubmarineLandslide::SubmarineLandslide(GlobalData *_gdata) :
	XProblem(_gdata),
	// boolean value to fill the chute with with water
	m_fillWater(get_option("m_fillWater", true)),
	// boolean value to a "Tree" (cylinder) in the middle of the chute
	m_addTree(get_option("tree", false)),
	// boolean value to add three "Pillars" (cylinder) at the end of the chute
	m_addPillars(get_option("pillars", false))
{
	m_name = "SubmarineLandslide";

	setFramework();

	const int mlsIters = get_option("mls", 0); // --mls N to enable MLS filter every N iterations
	const int ppH = get_option("ppH", 32); // --ppH N to change deltap to H/N

	if (mlsIters > 0) addFilter(MLS_FILTER, mlsIters);

	set_deltap(m_bulkWidth/ppH);

	m_depth = round_up(m_depth, m_deltap);

	// SPH parameters
	setSPHParameters();

	// Physical parameters
	setPhysicalParameters();

	// Drawing and saving times
	add_writer(VTKWRITER, 0.01);

	// Building the geometry
	buildGeometry();
}

void SubmarineLandslide::setFramework()
{
	// density diffusion terms, see DensityDiffusionType
	const int	rhodiff = get_option("density-diffusion", 1);
	const bool	newtonian = get_option("newtonian", false);

	SETUP_FRAMEWORK(
		rheology<PAPANASTASIOU>,
		turbulence_model<LAMINAR_FLOW>,
		visc_model<MORRIS>,
		visc_average<HARMONIC>,
		computational_visc<DYNAMIC>,
		boundary<DYN_BOUNDARY>,
		periodicity<PERIODIC_Y>,
		add_flags<ENABLE_MULTIFLUID>
	).select_options(
		newtonian, rheology<NEWTONIAN>()
	);
}

void SubmarineLandslide::setSPHParameters()
{
	simparams()->buildneibsfreq = 10;
	simparams()->tend = 5.0f;
}

void SubmarineLandslide::setPhysicalParameters()
{
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	const float g = length(physparams()->gravity);
	const float maxvel = sqrt(2*g*1);
	// purely for cosmetic reason, let's round the soundspeed to the next integer
	const float c0 = ceil(10*maxvel);

	m_sand = add_fluid(m_sandDensity);
	set_equation_of_state(m_sand, 7.0f, c0);
	set_dynamic_visc(m_sand, 1.f);
	set_yield_strength(m_sand, m_sandYieldStrength);

	m_water = add_fluid(m_waterDensity);
	set_equation_of_state(m_water, 7.0f, c0);
	set_dynamic_visc(m_water, 1.e-3f);
	set_yield_strength(m_water, m_waterYieldStrength);

	physparams()->dcoeff = 5.0f*g*1;
	physparams()->r0 = m_deltap;
}

bool SubmarineLandslide::is_sand(double4 particlePosition)
{
	double bulkX = particlePosition.x - m_chuteUpperWidth;
	double bulkZ = particlePosition.z; //- ( m_chuteHeight - m_bulkHeight );
	return ( bulkX > 0  && bulkX <= m_bulkWidth && bulkZ <= m_chuteHeight );
}

void SubmarineLandslide::initializeParticles(BufferList &buffers, const uint numParticles)
{
	// 1. warn the user if this is expected to take much time
	printf("Initializing particles density and mass...\n");

	// 2. grab the particle arrays from the buffer list
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();
	double4 *pos_global = buffers.getData<BUFFER_POS_GLOBAL>();
	float4 *pos = buffers.getData<BUFFER_POS>();

	// 3. iterate on the particles
	const float z_intf = m_chuteHeight;
	const float z_freeSurface = m_fillWater ? m_waterBoxHeight : z_intf;
	// pressure at interface, from heavy fluid
	const float g = length(physparams()->gravity);

	for (uint i = 0; i < numParticles; i++) {
		float rho = 1;
		double depth = z_freeSurface - pos_global[i].z;
		// for boundary particles, we use the density of sand,
		// fluid particles will override fluid_idx depending on whether they are water or sand
		int fluid_idx = m_sand;
		if (FLUID(info[i])) {
			fluid_idx = is_sand(pos_global[i]) ? m_sand : m_water;
			// hydrostatic density: for the heavy fluid, this is simply computed
			// as the density that gives pressure rho g h, with h depth
			rho = hydrostatic_density(depth, fluid_idx);
			// more complex way:
			if (fluid_idx == m_sand) {
				float P = physparams()->rho0[m_water]*(m_waterBoxHeight-z_intf)*g;
				// plus hydrostatic pressure from _our_ fluid
				P += physparams()->rho0[m_sand]*(z_intf - pos_global[i].z)*g;
				rho = density_for_pressure(P, m_sand);
			}
			info[i]= make_particleinfo(PT_FLUID, fluid_idx, i);
		} else if (BOUNDARY(info[i])) {
			rho = hydrostatic_density(depth, fluid_idx);
			info[i]= make_particleinfo(PT_BOUNDARY, fluid_idx, i);
		}
		// fix up the particle mass according to the actual density
		pos[i].w *= physical_density(rho, fluid_idx);
		vel[i].w = rho;
	}	
}

void SubmarineLandslide::buildGeometry()
{
	setPositioning(PP_CORNER);

	GeometryID obliqueChute = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(0.f, 0.f, 0.f),
		m_chuteWidth, m_depth, m_chuteHeight);

	// This plane is used to erase unnecessary fixed boundary particles
	GeometryID optimizationPlane = addPlane(1.f, 0.f, 1.f, -m_chuteWidth + m_chuteThickness, FT_UNFILL);
		setIntersectionType(optimizationPlane, IT_INTERSECT);

	GeometryID horizontalChute = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(0.f, 0.f, 0.f),
		m_chuteUpperWidth - m_gapFixOffset2, m_depth, m_chuteHeight);

	// This box is used to erase unnecessary fixed boundary particles
	GeometryID optimizationBox = addBox(GT_FIXED_BOUNDARY, FT_NOFILL, Point(0.f, 0.f, 0.f),
		m_chuteUpperWidth - m_gapFixOffset2, m_depth, m_chuteHeight - m_chuteThickness + 0.025f);

	GeometryID obliquePlane = addPlane(1.f, 0.f, 1.f, -m_chuteWidth - m_gapFixOffset, FT_UNFILL);
	setIntersectionType(obliquePlane, IT_SUBTRACT);

	GeometryID baseBoundary = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_chuteWidth - 0.1f, 0.f, -0.125f),
		m_waterBoxWidth - m_chuteWidth + 0.2f, m_depth, 0.1f);

	GeometryID leftBoundary = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(-0.125f, 0.f, m_chuteHeight - 0.1f),
		0.1f, m_depth, 0.2f);
	
	GeometryID rightBoundary = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_waterBoxWidth + 0.025f, 0.f, 0.f),
		0.1f, m_depth, m_waterBoxHeight);

	if(m_fillWater)
	{
		GeometryID waterBox = addBox(GT_FLUID, FT_SOLID, Point(0.f, 0.f, 0.f),
			m_waterBoxWidth, m_depth, m_waterBoxHeight);
	}
	
	Point bulkPoint = Point(m_chuteUpperWidth, 0.f, m_chuteHeight - m_bulkHeight);
	GeometryID bulk = addBox(GT_FLUID, FT_SOLID, bulkPoint,
		m_bulkWidth, m_depth, m_bulkHeight);
	
	GeometryID obliquePlane2 = addPlane(1.f, 0.f, 1.f, -m_chuteWidth + m_gapFixOffset, FT_UNFILL);
	setEraseOperation(obliquePlane2, ET_ERASE_FLUID);
	setIntersectionType(obliquePlane2, IT_INTERSECT);

	if(m_fillWater)
	{
		GeometryID fixWaterBox = addBox(GT_FLUID, FT_SOLID, Point(0.f, 0.f, m_chuteHeight + 0.025),
			m_waterBoxWidth, m_depth, m_waterBoxHeight - m_chuteHeight - m_gapFixOffset);
	}

	setPositioning(PP_CENTER);
	
	if(m_addTree){
		GeometryID tree = addCylinder(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_chuteWidth, m_depth/2.f, m_waterBoxHeight/2.f), 0.1f, m_waterBoxHeight);
	}

	if(m_addPillars)
	{
		GeometryID leftPillar = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_chuteWidth + 1.f, m_depth/3.f, m_waterBoxHeight/2.f), 0.1f, 0.1f, m_waterBoxHeight);
		GeometryID rightPillar = addBox(GT_FIXED_BOUNDARY, FT_SOLID, Point(m_chuteWidth + 1.f, 2*m_depth/3.f, m_waterBoxHeight/2.f), 0.1f, 0.1f, m_waterBoxHeight);
	}

}
